#include "cuSZp_entry.h"
#include "cuSZp_timer.h"
#include "cuSZp_utility.h"
#include "cuSZp.h"


extern "C"{
    /** Before entering SZp_compress, must allocate on device:
     * - d_cmpBytes
    */
    unsigned char* cuSZp_device_compress(float *oriData, size_t *outSize, float absErrBound, size_t nbEle){
        unsigned char *d_cmpBytes, *d_finalCmpBytes;
        hipStream_t stream;
        hipStreamCreate(&stream);
        hipMalloc((void**)&d_cmpBytes, sizeof(float)*nbEle);
        SZp_compress_deviceptr(oriData, d_cmpBytes, nbEle, outSize, absErrBound, stream);
        hipMalloc((void**)&d_finalCmpBytes, *outSize);
        hipMemcpy(d_finalCmpBytes, d_cmpBytes, *outSize, hipMemcpyDeviceToDevice);
        hipFree(d_cmpBytes);
	//hipFree(oriData);
        return d_finalCmpBytes;
    }

    /** Before entering SZp_decompress, must allocate on device:
     * - d_decData
    */
    float* cuSZp_device_decompress(size_t nbEle, unsigned char* cmpBytes, size_t cmpSize, float errorBound){
        float *d_decData;
        hipStream_t stream;
        hipStreamCreate(&stream);
        hipMalloc((void**)&d_decData, sizeof(float)*nbEle);
        SZp_decompress_deviceptr(d_decData, cmpBytes, nbEle, cmpSize, errorBound, stream);
        hipFree(cmpBytes);
	return d_decData;
    }
    
}
