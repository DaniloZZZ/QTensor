#include "cuSZp_entry.h"
#include "cuSZp.h"

void SZp_compress_hostptr(float* oriData, unsigned char* cmpBytes, size_t nbEle, size_t* cmpSize, float errorBound)
{
    // Data blocking.
    int bsize = cmp_tblock_size;
    int gsize = (nbEle + bsize * cmp_chunk - 1) / (bsize * cmp_chunk);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk;

    // Initializing global memory for GPU compression.
    float* d_oriData;
    unsigned char* d_cmpData;
    unsigned int* d_cmpOffset;
    int* d_flag;
    unsigned int glob_sync;
    hipMalloc((void**)&d_oriData, sizeof(float)*pad_nbEle);
    hipMemcpy(d_oriData, oriData, sizeof(float)*pad_nbEle, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_cmpData, sizeof(float)*pad_nbEle);
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_compress_kernel<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_cmpData, d_cmpOffset, d_flag, errorBound, nbEle);

    // Obtain compression ratio and move data back to CPU.  
    hipMemcpy(&glob_sync, d_cmpOffset+cmpOffSize-2, sizeof(unsigned int), hipMemcpyDeviceToHost);
    *cmpSize = (size_t)glob_sync + (nbEle+31)/32;
    hipMemcpy(cmpBytes, d_cmpData, *cmpSize*sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free memory that is used.
    hipFree(d_oriData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);
}


void SZp_decompress_hostptr(float* decData, unsigned char* cmpBytes, size_t nbEle, size_t cmpSize, float errorBound)
{
    // Data blocking.
    int bsize = dec_tblock_size;
    int gsize = (nbEle + bsize * dec_chunk - 1) / (bsize * dec_chunk);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * dec_chunk;

    // Initializing global memory for GPU compression.
    float* d_decData;
    unsigned char* d_cmpData;
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMalloc((void**)&d_decData, sizeof(float)*pad_nbEle);
    hipMemset(d_decData, 0, sizeof(float)*pad_nbEle);
    hipMalloc((void**)&d_cmpData, sizeof(float)*pad_nbEle);
    hipMemcpy(d_cmpData, cmpBytes, sizeof(unsigned char)*cmpSize, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_decompress_kernel<<<gridSize, blockSize, 0, stream>>>(d_decData, d_cmpData, d_cmpOffset, d_flag, errorBound, nbEle);

    // Move data back to CPU.
    hipMemcpy(decData, d_decData, sizeof(float)*nbEle, hipMemcpyDeviceToHost);

    // Free memoy that is used.
    hipFree(d_decData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);
}


void SZp_compress_deviceptr(float* d_oriData, unsigned char* d_cmpBytes, size_t nbEle, size_t* cmpSize, float errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = cmp_tblock_size;
    int gsize = (nbEle + bsize * cmp_chunk - 1) / (bsize * cmp_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    int* d_flag;
    unsigned int glob_sync;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_compress_kernel<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_flag, errorBound, nbEle);

    // Obtain compression ratio and move data back to CPU.  
    hipMemcpy(&glob_sync, d_cmpOffset+cmpOffSize-2, sizeof(unsigned int), hipMemcpyDeviceToHost);
    *cmpSize = (size_t)glob_sync + (nbEle+31)/32;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_flag);
}


void SZp_decompress_deviceptr(float* d_decData, unsigned char* d_cmpBytes, size_t nbEle, size_t cmpSize, float errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = dec_tblock_size;
    int gsize = (nbEle + bsize * dec_chunk - 1) / (bsize * dec_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);
    hipMemset(d_decData, 0, sizeof(float)*nbEle);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_decompress_kernel<<<gridSize, blockSize, 0, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_flag, errorBound, nbEle);
    
    // Free memoy that is used.
    hipFree(d_cmpOffset);
    hipFree(d_flag);
}