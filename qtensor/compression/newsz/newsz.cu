#include "hip/hip_runtime.h"
#include <stdio.h>
#include "newsz.h"
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <hipcub/hipcub.hpp>
// #include "cuCompactor.cuh"

#include "nvcomp/lz4.hpp"
#include "nvcomp.hpp"
#include "nvcomp/nvcompManagerFactory.hpp"

#define BLKS 40
#define THDS 128
#define FULL_MASK 0xffffffff

__device__ int g_ints;

struct int_predicate
{
    
	__host__ __device__
	bool operator()(const int x)
	{
		return x>0;
	}
};

struct to_copy
{
  __host__ __device__
  bool operator()(const uint8_t x)
  {
    return x==1;
  }
};




__global__ void compress(float *data, float *scales, float *zeropts, int8_t *out){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ float scratchpad[];
    __shared__ float min;
    __shared__ float max;

    typedef hipcub::BlockReduce<float, THDS> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage1;

    float item = data[blockIdx.x*blockDim.x+threadIdx.x];

    float tmax = BlockReduce(temp_storage1).Reduce(item, hipcub::Max());
    float tmin = BlockReduce(temp_storage1).Reduce(item, hipcub::Min());
    
    if (threadIdx.x==0)
    {
        max = tmax;
        min = tmin;
    }

    __syncthreads();

    float vrange = max - min;
    float scale = vrange/((2^8) - 1);
    int zeropt = -1*lrintf(min*scale) - (2^7);

    int q_item = lrintf(item/scale) + zeropt;

    // Clamp quantized value
    if(q_item>127)q_item = 127;
    if(q_item <-128)q_item = -128;
    int8_t q_val = (int8_t)(0xff & q_item);
    out[blockIdx.x*blockDim.x+threadIdx.x] = q_val;
    if (threadIdx.x==0)
    {
        scales[blockIdx.x] = scale;
        zeropts[blockIdx.x]= zeropt;
    }
    
}

__global__ void decompress(int8_t *q_data, float *scales, float *zeropts, float *out){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ float scratchpad[];
    __shared__ float min;
    __shared__ float max;

    typedef hipcub::BlockReduce<float, THDS> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage1;

    int8_t q_val = q_data[blockIdx.x*blockDim.x+threadIdx.x];

    out[blockIdx.x*blockDim.x+threadIdx.x] = (q_val - zeropts[bid])*scales[bid];
}

__global__ void p_ints(){
	printf("codebook entries used: %d\n", g_ints);
}

unsigned char* SZ_device_compress(float *data, size_t num_elements, int blocksize, size_t *outsize){
    float *scales, *zeropts;
    int8_t *q_out;
    unsigned char *cmpbytes;
    int num_blocks = num_elements/blocksize;

    hipMalloc(&scales, sizeof(float)*num_blocks);
    hipMalloc(&zeropts,sizeof(float)*num_blocks);
    hipMalloc(&q_out, num_elements);

    using namespace nvcomp;

    hipStream_t stream;
    hipStreamCreate(&stream);

    const int chunk_size = 1 << 16;
    nvcompType_t data_type = NVCOMP_TYPE_CHAR;

     

    compress<<<num_blocks, blocksize>>>(data, scales, zeropts, q_out);
    hipDeviceSynchronize();

    LZ4Manager nvcomp_manager{chunk_size, data_type, stream};
    CompressionConfig comp_config = nvcomp_manager.configure_compression(num_elements);

    uint8_t* comp_buffer;
    hipMalloc(&comp_buffer, comp_config.max_compressed_buffer_size);
    
    nvcomp_manager.compress((const uint8_t *)q_out, comp_buffer, comp_config);

    size_t c_size = nvcomp_manager.get_compressed_output_size(comp_buffer);
    hipFree(q_out);

    *outsize = sizeof(float)*(num_blocks+num_blocks)+c_size;
    hipMalloc(&cmpbytes, *outsize);

    hipMemcpy(cmpbytes, (unsigned char *)scales, sizeof(float)*num_blocks, hipMemcpyDeviceToDevice);
    hipMemcpy(cmpbytes+sizeof(float)*num_blocks, (unsigned char *)zeropts, sizeof(float)*num_blocks, hipMemcpyDeviceToDevice);
    hipMemcpy(cmpbytes+sizeof(float)*num_blocks+sizeof(float)*num_blocks, comp_buffer, c_size, hipMemcpyDeviceToDevice);

    float h_firstscale;
    hipMemcpy(&h_firstscale, cmpbytes, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(scales);
    hipFree(zeropts);
    hipFree(comp_buffer);
    return cmpbytes;
}

float* SZ_device_decompress(unsigned char *cmpbytes, size_t num_elements, int blocksize, size_t *cmpsize){
    float *scales, *zeropts;
    uint8_t *q_cmp;
    int8_t *q_vals;
    float *out;
    int num_blocks = num_elements/blocksize;
    size_t c_size = *cmpsize-(2*sizeof(float)*num_blocks);

    float first_val, *d_first;

    hipMalloc(&d_first, sizeof(float));
    hipMemcpy((unsigned char *)&first_val, cmpbytes, sizeof(float), hipMemcpyDeviceToHost);



    hipMalloc((void **)&scales, sizeof(float)*num_blocks);
    hipMalloc((void **)&zeropts,sizeof(float)*num_blocks);
    hipMalloc((void **)&q_cmp, c_size);
    hipMemcpy((unsigned char *)scales, cmpbytes, sizeof(float)*num_blocks, hipMemcpyDeviceToDevice);
    
    hipMemcpy((unsigned char *)zeropts, cmpbytes+sizeof(float)*num_blocks, sizeof(float)*num_blocks, hipMemcpyDeviceToDevice);
    
    hipMemcpy(q_cmp, cmpbytes+sizeof(float)*num_blocks+sizeof(float)*num_blocks, c_size, hipMemcpyDeviceToDevice);
    hipStream_t stream;
    hipStreamCreate(&stream);

    const int chunk_size = 1 << 16;
    
    
    nvcompType_t data_type = NVCOMP_TYPE_CHAR;

    auto decomp_manager = nvcomp::create_manager(q_cmp, stream);

    nvcomp::DecompressionConfig decomp_config = decomp_manager->configure_decompression((uint8_t *)q_cmp);
    hipMalloc(&q_vals, num_elements);

    decomp_manager->decompress((uint8_t*)q_vals, (uint8_t*)q_cmp, decomp_config);
    hipFree(q_cmp);

    hipMalloc(&out, sizeof(float)*num_elements);

    decompress<<<num_blocks, blocksize>>>(q_vals, scales, zeropts, out);
    hipDeviceSynchronize();
    
    hipFree(scales);
    hipFree(zeropts);
    hipFree(q_vals);

    return out;
}

int main(int argc, char** argv){
    char oriFilePath[640], outputFilePath[645];
    float* data;
    size_t nbEle;
    if(argc < 3)
    {
		printf("Usage: testfloat_compress_fastmode2 [srcFilePath] [block size] [err bound] [--cuda]\n");
		printf("Example: testfloat_compress_fastmode2 testfloat_8_8_128.dat 64 1E-3 --cuda\n");
		exit(0);
    }

    sprintf(oriFilePath, "%s", argv[1]);
    int blockSize = atoi(argv[2]);
    float errBound = atof(argv[3]);
    nbEle = atoi(argv[4]);

    data = (float*)malloc(sizeof(float)*nbEle);
    sprintf(outputFilePath, "%s.sznew", oriFilePath);

    FILE *in_file;
    in_file = fopen(oriFilePath, "rb");
    
    fread(data, sizeof(float), nbEle, in_file);
    fclose(in_file);
    
    float max = data[0];
    float min = data[0];
    for(int i=0;i<nbEle;i++){
	if(data[i]>=max){
		max = data[i];
	}
	if(data[i]<=min){
		min = data[i];
	}
    }
    errBound = errBound*(max-min);

    // Move to device
    float *d_data;
    unsigned char *cmpbytes;
    size_t outsize;
    hipMalloc(&d_data, sizeof(float)*nbEle);
    hipMemcpy(d_data, data, sizeof(float)*nbEle, hipMemcpyHostToDevice);
    //SZ_device_compress(d_data, nbEle, errBound, blockSize, cmpbytes, &outsize);

    hipFree(d_data);
    
}
